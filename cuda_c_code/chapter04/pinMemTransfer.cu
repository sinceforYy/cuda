#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
* Host memory is allocated using hipHostMalloc to create a page-locked host array.
*/
int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    CHECK(hipSetDevice(dev));

    // memory size
    unsigned int isize = 1 << 22;
    unsigned int nbytes = isize * sizeof(float);

    // get device information
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp,dev));

    if (!deviceProp.canMapHostMemory)
    {
        printf("Device %d does not support mapping CPU host memory!\n", dev);
        CHECK(hipDeviceReset());
        exit(EXIT_SUCCESS);
    }

    printf("%s starting at ", argv[0]);
    printf("device %d: %s memory size %d nbyte %5.2f MB canMap %d\n", dev,
        deviceProp.name, isize, nbytes / (1024.0f * 1024.0f),
        deviceProp.canMapHostMemory);
    
    // allocate pinned host mmeory
    float *h_a;
    CHECK(hipHostMalloc((float **)&h_a, nbytes));

    // allocate device memory
    float *d_a;
    CHECK(hipMalloc((float **)&d_a, nbytes));

    // initialize host memory
    memset(h_a, 0, nbytes);

    for (int i = 0; i < isize; i++) h_a[i] = 100.10f;

    // transfer data from the host to the device
    CHECK(hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice));

    // transfer data from the deivce to the host
    CHECK(hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost));

    // free memory
    CHECK(hipFree(d_a));
    CHECK(hipHostFree(h_a));

    // reset device
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}